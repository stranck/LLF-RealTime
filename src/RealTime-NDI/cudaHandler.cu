#include "hip/hip_runtime.h"
#include "cudaHandler.cuh"

#include "../CUDA/cuda.cuh"
#include <cuda/semaphore>

cuda::counting_semaphore<cuda::thread_scope_system, 1> frameAvailable;
cuda::counting_semaphore<cuda::thread_scope_system> hostSemaphore;
volatile uint32_t widthIn = 0, heightIn = 0, widthOut = 0, heightOut = 0;
uint64_t lastDeviceBufferDimension = 0;
uint64_t lastHostBufferDimension = 0;
CUDAbuffers *cudaBuffers;
Pixel4u8 *hostN2Tbuffer;
Pixel4u8 *hostT2Nbuffer;
Image3 *workingImage;

float _sigma, _alpha, _beta;
uint32_t _nThreads, _nBlocks;
uint8_t _nLevels;

#ifdef ON_WINDOWS
	LPDWORD gpuProcessingTID;
	DWORD WINAPI gpuProcessingThread_entryPoint(LPVOID lpParameter){ gpuProcessingThread(); return 0; }
#else
	pthread_t gpuProcessingTID;
	void * gpuProcessingThread_entryPoint(void *param){ gpuProcessingThread(); return NULL; }
#endif
void startGpuProcessingThread(float sigma, float alpha, float beta, uint8_t nLevels, uint32_t nThreads, uint32_t nBlocks){
	_sigma = sigma;
	_alpha = alpha;
	_beta = beta;
	_nLevels = nLevels;
	_nThreads = nThreads;
	_nBlocks = nBlocks;
	#ifdef ON_WINDOWS
		CreateThread(NULL, 0, gpuProcessingThread_entryPoint, NULL, 0, gpuProcessingTID);
	#else
		pthread_create(&gpuProcessingTID, 0, gpuProcessingThread_entryPoint, NULL);
	#endif
}

void handleIncomingFrame(NDIlib_video_frame_v2_t *ndiVideoFrame){
	hostSemaphore.acquire();
	widthIn = ndiVideoFrame->xres;
	heightIn = ndiVideoFrame->yres;
	uint64_t frameDimension = widthIn * heightIn;
	uint64_t frameDimensionBytes = frameDimension * sizeof(Pixel4u8);
	if(frameDimension > lastHostBufferDimension){ //so we don't reduce the size before we output the rendered frame
		free(hostN2Tbuffer);
		free(hostT2Nbuffer);
		hostN2Tbuffer = (Pixel4u8 *) malloc(frameDimensionBytes);
		hostT2Nbuffer = (Pixel4u8 *) malloc(frameDimensionBytes);
		lastHostBufferDimension = frameDimension;
	}
	memcpy(hostN2Tbuffer, ndiVideoFrame->p_data, frameDimensionBytes);
	hostSemaphore.release();
	frameAvailable.release();
}
void writeOutputFrame(NDIlib_video_frame_v2_t *ndiVideoFrame){
	hostSemaphore.acquire();
	uint64_t frameDimensionBytes = lastHostBufferDimension * sizeof(Pixel4u8);
	ndiVideoFrame->xres = min(ndiVideoFrame->xres, widthOut);
	ndiVideoFrame->yres = min(ndiVideoFrame->yres, heightOut);
	uint64_t outFrameDim = widthOut * heightOut * sizeof(Pixel4u8);
	frameDimensionBytes = min(frameDimensionBytes, outFrameDim);
	memcpy(ndiVideoFrame->p_data, hostT2Nbuffer, frameDimensionBytes);
	hostSemaphore.release();
}


void initProcessingThread(){
	hostN2Tbuffer = (Pixel4u8 *) malloc(1);
	hostT2Nbuffer = (Pixel4u8 *) malloc(1);
	hostSemaphore.release();

	workingImage = makeImage3(1, 1);

	cudaBuffers = (CUDAbuffers *) malloc(sizeof(CUDAbuffers));
	initCUDAbuffers(cudaBuffers, 200, 200, _nLevels);
}
void gpuProcessingThread(){
	initProcessingThread();
	while(true){
		frameAvailable.acquire(); //Wait for an available frame

		hostSemaphore.acquire(); //Copies the image locally
		workingImage->width = widthIn;
		workingImage->height = heightIn;
		uint32_t dim = widthIn * heightIn;
		if(dim > lastDeviceBufferDimension){
			destroyImage3(&workingImage);
			destroyCUDAbuffers(cudaBuffers, _nLevels);
			initCUDAbuffers(cudaBuffers, widthIn, heightIn, _nLevels);
			workingImage = makeImage3(widthIn, heightIn);
			lastDeviceBufferDimension = dim;
		}
		Pixel3 *pxs = workingImage->pixels;
		for(uint32_t i = 0; i < dim; i++){
			pxs[i].x = hostN2Tbuffer[i].x / 255.0f;
			pxs[i].y = hostN2Tbuffer[i].y / 255.0f;
			pxs[i].z = hostN2Tbuffer[i].z / 255.0f;
		}
		hostSemaphore.release();
	
		llf(workingImage, _sigma, _alpha, _beta, _nLevels, _nThreads, _nBlocks, cudaBuffers);

		hostSemaphore.acquire();
		widthOut = workingImage->width;
		heightOut = workingImage->height;
		for(uint32_t i = 0; i < dim; i++){
			hostT2Nbuffer[i].x = roundfu8(255.0f * pxs[i].x);
			hostT2Nbuffer[i].y = roundfu8(255.0f * pxs[i].y);
			hostT2Nbuffer[i].z = roundfu8(255.0f * pxs[i].z);
			hostT2Nbuffer[i].w = 0xff;
		}
		hostSemaphore.release();
	}
}