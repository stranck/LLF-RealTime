#include "hip/hip_runtime.h"
#include "cudaUtils.cuh"

__host__ Kernel createFilterDevice(){
	const float params[KERNEL_DIMENSION] = {0.05, 0.25, 0.4, 0.25, 0.05};
	float h_filter[KERNEL_DIMENSION * KERNEL_DIMENSION];

	for(uint8_t i = 0; i < KERNEL_DIMENSION; i++){
		for(uint8_t j = 0; j < KERNEL_DIMENSION; j++){
			h_filter[getKernelPosition(i, j)] = params[i] * params[j];
		}
	}

	Kernel d_filter;
	CHECK(hipMalloc((void**) &d_filter, KERNEL_DIMENSION * KERNEL_DIMENSION * sizeof(float)));
	printff("D_FILTER ADDR: 0x%016llx\n", d_filter);
	CHECK(hipMemcpy(d_filter, h_filter, KERNEL_DIMENSION * KERNEL_DIMENSION * sizeof(float), hipMemcpyHostToDevice));
	return d_filter;
}
__host__ void destroyFilterDevice(Kernel d_k){
	CHECK(hipFree(d_k));
}

__device__ Pyramid d_createPyramid(uint32_t width, uint32_t height, uint8_t nLevels){
	nLevels++; //Pyramids has one more layer!
	Pyramid p;
	hipMalloc(&p, nLevels * sizeof(Image3*));
	printf("d_createPyramid: Dimensions: %03dx%03d @ %d levels    Pyramid at 0x%012llx\n", width, height, nLevels, p);
	for(uint8_t i = 0; i < nLevels; i++){
		p[i] = d_makeImage3(width, height);
		width = width / 2 + (width & 1);
		height = height / 2 + (height & 1);
	}
	return p;
}
__host__ Pyramid createPyramidDevice(uint32_t width, uint32_t height, uint8_t nLevels){
	nLevels++; //Pyramids has one more layer!
	Pyramid h_p = (Pyramid) allocStack(nLevels * sizeof(Image3*));
	for(uint8_t i = 0; i < nLevels; i++){
		h_p[i] = makeImage3Device(width, height);
		width = width / 2 + (width & 1);
		height = height / 2 + (height & 1);
	}

	Pyramid d_p;
	CHECK(hipMalloc((void**) &d_p, nLevels * sizeof(Image3*)));
	printff("CreatePyramidDevice: malloc pyramid at 0x%032llx. Params: %u\n", d_p, nLevels);
	CHECK(hipMemcpy(d_p, h_p, nLevels * sizeof(Image3*), hipMemcpyHostToDevice));
	return d_p;
}
__device__ void d_destroydPyramid(Pyramid pyr, uint8_t nLevels){
	for(uint8_t i = 0; i <= nLevels; i++)
		d_destroyImage3(pyr[i]);
	hipFree(pyr);
}
__host__ void destroyPyramidDevice(Pyramid d_pyr, uint8_t h_nLevels){
	Pyramid h_pyr = (Pyramid) allocStack((h_nLevels + 1)* sizeof(Image3*));
	CHECK(hipMemcpy(h_pyr, d_pyr, (h_nLevels + 1) * sizeof(Image3*), hipMemcpyDeviceToHost));
	for(uint8_t i = 0; i <= h_nLevels; i++)
		destroyImage3Device(h_pyr[i]);
	CHECK(hipFree(d_pyr));
}

__device__ Image3 * d_makeImage3(uint32_t width, uint32_t height){
	if(blockIdx.x == 0 && threadIdx.x == 0){
		Image3 *i;
		Pixel3 *img;
		hipError_t errImg = hipMalloc(&i, sizeof(Image3));
		hipError_t errPx = hipMalloc(&img, width * height * sizeof(Pixel3));
		i -> width = width;
		i -> height = height;
		i -> pixels = img;
		printf("d_makeImage3: Dimensions: % 3dx% 3d    Pixels at 0x%012llx    Image3 at 0x%012llx    Error img: %s     Error pxs: %s\n", width, height, i, img, hipGetErrorString(errImg), hipGetErrorString(errPx));
		return i;
	}
}
__host__ Image3 * makeImage3Device(uint32_t width, uint32_t height){
	Pixel3 *d_img;
	CHECK(hipMalloc((void**) &d_img, width * height * sizeof(Pixel3)));
	Image3 h_i;
	h_i.width = width;
	h_i.height = height;
	h_i.pixels = d_img;

	Image3 *d_i;
	CHECK(hipMalloc((void**) &d_i, sizeof(Image3)));
	CHECK(hipMemcpy(d_i, &h_i, sizeof(Image3), hipMemcpyHostToDevice));
	return d_i;
}
__device__ void d_destroyImage3(Image3 *img){
	hipFree(img -> pixels);
	hipFree(img);
}
__host__ void destroyImage3Device(Image3 *d_img){
	Image3 h_img;
	CHECK(hipMemcpy(&h_img, d_img, sizeof(Image3), hipMemcpyDeviceToHost));
	CHECK(hipFree(h_img.pixels));
	CHECK(hipFree(d_img));
}

__host__ void copyImg3Host2Device(Image3 *d_imgDst, Image3 *h_imgSrc){
	Image3 h_i;
	CHECK(hipMemcpy(&h_i, d_imgDst, sizeof(Image3), hipMemcpyDeviceToHost));
	h_i.width = h_imgSrc->width;
	h_i.height = h_imgSrc->height;
	CHECK(hipMemcpy(d_imgDst, &h_i, sizeof(Image3), hipMemcpyHostToDevice));
	CHECK(hipMemcpy(h_i.pixels, h_imgSrc->pixels, h_imgSrc->width * h_imgSrc->height * sizeof(Pixel3), hipMemcpyHostToDevice));
}
__host__ void copyImg3Device2Host(Image3 *h_imgDst, Image3 *d_imgSrc){
	Image3 h_i;
	CHECK(hipMemcpy(&h_i, d_imgSrc, sizeof(Image3), hipMemcpyDeviceToHost));
	h_imgDst->width = h_i.width;
	h_imgDst->height = h_i.height;
	size_t h_toCopy = (h_i.width) * (h_i.height) * sizeof(Pixel3);
	CHECK(hipMemcpy(h_imgDst->pixels, h_i.pixels, h_toCopy, hipMemcpyDeviceToHost));
}
__device__ void d_imgcpy3(Image3 *d_dest, Image3 *d_source){
	__shared__ uint32_t dim;
	__shared__ Pixel3* d_destPxs;
	__shared__ Pixel3* d_srcPxs;

	if(threadIdx.x == 0){
		d_dest->width = d_source->width;
		d_dest->height = d_source->height;
		dim = d_dest->width * d_dest->height;
		d_destPxs = d_dest->pixels;
		d_srcPxs = d_source->pixels;
	}
	__syncthreads();
	uint32_t max = dim / blockDim.x;
	for(uint32_t i = 0; i <= max; i++){
		uint32_t idx = i * blockDim.x + threadIdx.x;
		if(idx < dim)
			d_destPxs[idx] = d_srcPxs[idx];
	}
	__syncthreads();
}
__global__ void d_copyPyrLevel(Pyramid dst_pyr, Pyramid src_pyr, uint8_t level){
	d_imgcpy3(dst_pyr[level], src_pyr[level]);
} 

__host__ Image3 * getImageFromPyramidDevice(Pyramid d_pyr, uint8_t h_level){
	Pyramid h_pyr = (Pyramid) allocStack((h_level + 1) * sizeof(Image3*)); //We just need to copy up to level pointers;
	CHECK(hipMemcpy(h_pyr, d_pyr, (h_level + 1) * sizeof(Image3*), hipMemcpyDeviceToHost));
	return h_pyr[h_level];
}
__host__ void getPyramidDimensionsAtLayer(Pyramid d_pyr, uint8_t h_level, uint32_t *h_width, uint32_t *h_height){
	Image3 h_lvl;
	Image3 *d_img = getImageFromPyramidDevice(d_pyr, h_level);
	CHECK(hipMemcpy(&h_lvl, d_img, sizeof(Image3), hipMemcpyDeviceToHost));
	*h_width = h_lvl.width;
	*h_height = h_lvl.height;
}

__device__ void d_subimage3Remap_shared(Pixel3 *destPx, Image3 *source, uint32_t startX, uint32_t endX, uint32_t startY, uint32_t endY, const Pixel3 g0, float sigma, float alpha, float beta){
	uint32_t w = endX - startX;
	uint32_t h = endY - startY;

	Pixel3 *srcPx = source->pixels;
	uint32_t srcW = source->width;
	uint32_t dim = w * h;
	uint32_t max = dim / blockDim.x;
	for(uint32_t i = 0; i <= max; i++){
		uint32_t idx = i * blockDim.x + threadIdx.x;
		if(idx < dim){
			uint32_t x = idx % w, y = idx / w;
			uint32_t finalY = startY + y;

			Pixel3 p = d_getPixel3(srcPx, srcW, startX + x, finalY);
			Pixel3 remapped = d_remapSinglePixel(p, g0, sigma, alpha, beta);
			d_setPixel3(destPx, w, x, y, remapped);
		}
	}
	__syncthreads();
}
__device__ void d_subimage3Remap(Image3 *dest, Image3 *source, uint32_t startX, uint32_t endX, uint32_t startY, uint32_t endY, const Pixel3 g0, float sigma, float alpha, float beta){
	uint32_t w = endX - startX;
	uint32_t h = endY - startY;
	if(threadIdx.x == 0){
		dest->width = w;
		dest->height = h;
	}

	Pixel3 *destPx = dest->pixels, *srcPx = source->pixels;
	uint32_t srcW = source->width;
	uint32_t dim = w * h;
	uint32_t max = dim / blockDim.x;
	for(uint32_t i = 0; i <= max; i++){
		uint32_t idx = i * blockDim.x + threadIdx.x;
		if(idx < dim){
			uint32_t x = idx % w, y = idx / w;
			uint32_t finalY = startY + y;

			Pixel3 p = d_getPixel3(srcPx, srcW, startX + x, finalY);
			Pixel3 remapped = d_remapSinglePixel(p, g0, sigma, alpha, beta);
			d_setPixel3(destPx, w, x, y, remapped);
		}
	}
	__syncthreads();
}
__device__ void d_subimage3(Image3 *dest, Image3 *source, uint32_t startX, uint32_t endX, uint32_t startY, uint32_t endY){
	uint32_t w = endX - startX;
	uint32_t h = endY - startY;
	if(threadIdx.x == 0){
		dest->width = w;
		dest->height = h;
	}

	Pixel3 *destPx = dest->pixels, *srcPx = source->pixels;
	uint32_t srcW = source->width;
	uint32_t dim = w * h;
	uint32_t max = dim / blockDim.x;
	for(uint32_t i = 0; i <= max; i++){
		uint32_t idx = i * blockDim.x + threadIdx.x;
		if(idx < dim){
			uint32_t x = idx % w, y = idx / w;
			uint32_t finalY = startY + y;
			d_setPixel3(destPx, w, x, y, d_getPixel3(srcPx, srcW, startX + x, finalY));
		}
	}
	__syncthreads();
}

__global__ void d_clampImage3(Image3 *img){
	__shared__ uint32_t dim;
	__shared__ Pixel3 *px;

	if(threadIdx.x == 0){
		dim = img->width * img->height;
		px = img->pixels;
	}
	__syncthreads();

	uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
	if(i < dim){
		px[i].x = d_clamp(px[i].x, 0, 1);
		px[i].y = d_clamp(px[i].y, 0, 1);
		px[i].z = d_clamp(px[i].z, 0, 1);
	}
	__syncthreads();
}
__device__ float d_clamp(float a, float min_, float max_) {
	int minFlag = a < min_;
	int maxFlag = a > max_;
	int flag = minFlag + maxFlag;
	//if(flag > 1) flag = 1; //no way they are both true at the same time IF THE PARAMS ARE CORRECT :<
	return a * (1 - flag) + min_ * minFlag + max_ * maxFlag;
}
__device__ float d_smoothstep(float a, float b, float u) {
	float t = d_clamp((u - a) / (b - a), 0.0, 1.0);
	return t * t * (3 - 2 * t);
}

__device__ inline Pixel3 d_remapSinglePixel(const Pixel3 source, const Pixel3 g0, float sigma, float alpha, float beta){
	Pixel3 delta;
	vec3Sub(delta, source, g0);
	float mag = sqrt(delta.x * delta.x + delta.y * delta.y + delta.z * delta.z);
	if(mag > 1e-10) vec3DivC(delta, delta, mag);

	int details = mag < sigma;
	float fraction = mag / sigma;
	float polynomial = pow(fraction, alpha);
	if(alpha < 1){ //alpha is one of the entire llf params, so ALL the threads will always take the same branch
		const float kNoiseLevel = 0.01;
		float blend = d_smoothstep(kNoiseLevel, 2 * kNoiseLevel, fraction * sigma);
		polynomial = blend * polynomial + (1 - blend) * fraction;
	}
	float d = (sigma * polynomial) * details + (((mag - sigma) * beta) + sigma) * (1 - details);
	vec3MulC(delta, delta, d);
	vec3Add(delta, g0, delta);
	return delta;
}
__device__ void d_remap(Image3 * img, const Pixel3 g0, float sigma, float alpha, float beta){
	uint32_t dim = img -> width * img -> height;
	uint32_t max = dim / blockDim.x;
	Pixel3 *pixels = img -> pixels;
	for(uint32_t i = 0; i <= max; i++){
		uint32_t idx = i * blockDim.x + threadIdx.x;
		if(idx < dim)
			pixels[idx] = d_remapSinglePixel(pixels[idx], g0, sigma, alpha, beta);
	}
	__syncthreads();
}