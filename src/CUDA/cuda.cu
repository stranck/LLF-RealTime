#include "hip/hip_runtime.h"
#include "cuda.cuh"

#include "../utils/test/testimage.h"
#include <sys/time.h>

__device__ Pixel3 upsampleConvolveSubtractSinglePixel(Image3 *source, Pixel3 gaussPx, Kernel kernel, uint32_t i, uint32_t j, Pixel3 *convolveWorkingBuffer){
	uint32_t smallWidth = source->width, smallHeight = source->height;
	Pixel3* srcPx = source->pixels;
	const int32_t  xstart = -1 * KERNEL_DIMENSION / 2;
	const int32_t  ystart = -1 * KERNEL_DIMENSION / 2;
	
	//__shared__ Pixel3 ups;
	//ups = zero3vect;
	Pixel3 ups = zero3vect;
	//for (uint32_t y = 0; y < KERNEL_DIMENSION; y++) {
	//	for (uint32_t x = 0; x < KERNEL_DIMENSION; x++) {
	uint32_t idx = threadIdx.x;
	if(idx < (KERNEL_DIMENSION * KERNEL_DIMENSION)){
		uint32_t x = idx % KERNEL_DIMENSION, y = idx / KERNEL_DIMENSION;

		int32_t jy = (j + ystart + y) / 2;
		int32_t ix = (i + xstart + x) / 2;

		int32_t oob = ix >= 0 && ix < smallWidth && jy >= 0 && jy < smallHeight;
		int32_t fi = ix * oob + (i / 2) * (1 - oob), fj = jy * oob + (j / 2) * (1 - oob);

		float kern_elem = kernel[getKernelPosition(x, y)];
		Pixel3 px = d_getPixel3(srcPx, smallWidth, fi, fj); //srcPx[fj * smallWidth + fi];

		convolveWorkingBuffer[idx] = vec3MulC(px, kern_elem, Pixel3);
		//ups.x += px.x * kern_elem;
		//ups.y += px.y * kern_elem;
		//ups.z += px.z * kern_elem;
		
		//float *upsAddr = (float *) &ups, *pxAddr = (float *) &px;
		//uint32_t addrId = idx % 3;
		//for(uint32_t n = 0; n < 3; n++){
		//	uint32_t addrIndex = ((addrId + n) % 3);
		//	atomicAdd(upsAddr + addrIndex, *(pxAddr + addrIndex) * kern_elem);
		//}
		//atomicAdd(&ups.x, px.x * kern_elem);
		//atomicAdd(&ups.y, px.y * kern_elem);
		//atomicAdd(&ups.z, px.z * kern_elem);
		//}
		for(uint32_t stride = KERNEL_DIMENSION * KERNEL_DIMENSION / 2; stride > 1; stride = stride >> 1){
			//__syncthreads();
			if(idx < stride){
				convolveWorkingBuffer[idx].x += convolveWorkingBuffer[idx + stride].x;
				convolveWorkingBuffer[idx].y += convolveWorkingBuffer[idx + stride].y;
				convolveWorkingBuffer[idx].z += convolveWorkingBuffer[idx + stride].z;
			}
		}
	}
	/*if(idx == 0){
		for(uint32_t n = 0; n < KERNEL_DIMENSION * KERNEL_DIMENSION; n++){
			ups.x += convolveWorkingBuffer[n].x;
			ups.y += convolveWorkingBuffer[n].y;
			ups.z += convolveWorkingBuffer[n].z;
		}
	}*/
	//__syncthreads();

	return vec3Sub(gaussPx, ups, Pixel3);
}
__device__ void upsampleConvolveSubtract_fast(Image3 *dest, Image3 *source, Image3 *currentGauss, Kernel kernel, Pixel3 *ds_upsampled){
	//printf("upsampleConvolve_fast addr:  DEST: 0x%012llx       SRC: 0x%012llx        GAUS: 0x%012llx\n", dest, source, currentGauss);
	uint32_t smallWidth = source->width, smallHeight = source->height;
	uint32_t uppedW = smallWidth << 1;
	uint32_t uppedH = smallHeight << 1;
	uint32_t currentGaussW = currentGauss->width;
	uint32_t yEnd = min(currentGauss->height, uppedH);
	Pixel3 *destPx = dest->pixels, *srcPx = source->pixels, *crtGssPx = currentGauss->pixels;
	if(threadIdx.x == 0){
		dest->width = uppedW;
		dest->height = uppedH;
	}
	//__syncthreads();
	uint32_t xEnd = min(currentGaussW, uppedW);
	//printf("upsampleConvolve_fast start    sw: %d    sh: %d    uw: %d    uh: %d    gw: %d    gh: %d    xe: %d    ye: %d\n", smallWidth, smallHeight, uppedW, uppedH, currentGaussW, currentGauss->height, xEnd, yEnd);
	const uint8_t  rows = KERNEL_DIMENSION;
	const uint8_t  cols = KERNEL_DIMENSION;
	const int32_t  xstart = -1 * cols / 2;
	const int32_t  ystart = -1 * rows / 2;
	
	uint32_t dim = smallWidth * smallHeight;
	uint32_t max = dim / blockDim.x;
	for(uint32_t i = 0; i <= max; i++){
		uint32_t idx = i * blockDim.x + threadIdx.x;
		if(idx < dim){
			uint32_t x = idx % smallWidth, y = idx / smallWidth;
			d_setPixel3(ds_upsampled, smallWidth, x, y, d_getPixel3(srcPx, smallWidth, x, y));
			//ds_upsampled[y * smallWidth + x] = *getPixel3(source, x, y);
		}
	}
	__syncthreads();


	dim = xEnd * yEnd;
	max = dim / blockDim.x;
	for(uint32_t li = 0; li <= max; li++){
		uint32_t idx = li * blockDim.x + threadIdx.x;
		if(idx < dim){
			uint32_t i = idx % xEnd, j = idx / xEnd;

			Pixel3 ups = zero3vect;
			for (uint32_t y = 0; y < rows; y++) {
                int32_t jy = (j + ystart + y) / 2;
				for (uint32_t x = 0; x < cols; x++) {
                    int32_t ix = (i + xstart + x) / 2;

					int32_t oob = ix >= 0 && ix < smallWidth && jy >= 0 && jy < smallHeight;
					int32_t fi = ix * oob + (i / 2) * (1 - oob), fj = jy * oob + (j / 2) * (1 - oob);

					float kern_elem = kernel[getKernelPosition(x, y)];
					Pixel3 px = d_getPixel3(ds_upsampled, smallWidth, fi, fj); //ds_upsampled[fj * uppedW + fi]; //*getPixel3(source, ix, jy);
					ups.x += px.x * kern_elem;
					ups.y += px.y * kern_elem;
					ups.z += px.z * kern_elem;
				}
			}

			Pixel3 crr = d_getPixel3(crtGssPx, currentGaussW, i, j);
			Pixel3 sub = vec3Sub(crr, ups, Pixel3);
			//printf("OrgDim: %dx%d \t \t CrrDim: %dx%d \t \t Writing at %dx%d\n", dest->originalW, dest->originalH, dest->width, dest->height, i, j);
			d_setPixel3(destPx, xEnd, i, j, sub);
		}
	}
	__syncthreads();
}

__device__ void downsampleConvolve_fast(Image3 *dest, Image3 *source, uint32_t *width, uint32_t *height, Kernel filter, Pixel3 *ds_downsampled){
	const uint32_t originalW = *width, originalH = *height;
	const uint32_t downW = originalW / 2, downH = originalH / 2;
	Pixel3 *srcPx = source->pixels;
	Pixel3 *dstPx = dest->pixels;
	*width = downW;
	*height = downH;
	if(threadIdx.x == 0){
		dest->width = downW;
		dest->height = downH;
	}
	//__syncthreads();
	uint32_t startingX = originalW & 1;
	uint32_t startingY = originalH & 1;
	
	uint32_t dim = downW * downH;
	uint32_t max = dim / blockDim.x;
	for(uint32_t i = 0; i <= max; i++){
		uint32_t idx = i * blockDim.x + threadIdx.x;

		if(idx < dim){
			uint32_t x = idx % downW, y = idx / downW;
			d_setPixel3(ds_downsampled, downW, x, y, d_getPixel3(srcPx, originalW, (x * 2) + startingX, (y * 2) + startingY));
			//ds_downsampled[y * lcl_width + x] = *getPixel3(source, (x * 2) - startingX, (y * 2) - startingY);
		}
	}
	__syncthreads();

	const uint8_t  rows = KERNEL_DIMENSION;
	const uint8_t  cols = KERNEL_DIMENSION;
	const int32_t  xstart = -1 * cols / 2;
	const int32_t  ystart = -1 * rows / 2;

	for(uint32_t li = 0; li <= max; li++){
		uint32_t idx = li * blockDim.x + threadIdx.x;

		if(idx < dim){
			uint32_t i = idx % downW, j = idx / downW;
			Pixel3 c = zero3vect;
			for (int32_t y = 0; y < rows; y++) {
				int32_t jy = j + ystart + y;
				for (int32_t x = 0; x < cols; x++) {
					int32_t ix = i + xstart + x;

					int32_t oob = ix >= 0 && ix < downW && jy >= 0 && jy < downH;
					int32_t fi = ix * oob + i * (1 - oob), fj = jy * oob + j * (1 - oob);

					float kern_elem = filter[getKernelPosition(x, y)];
					Pixel3 px = d_getPixel3(ds_downsampled, downW, fi, fj); //ds_downsampled[fj * lcl_width + fi]; //*getPixel3(source, fx, fj);
					c.x += px.x * kern_elem;
					c.y += px.y * kern_elem;
					c.z += px.z * kern_elem;
				}
			}
			d_setPixel3(dstPx, downW, i, j, c);
			//setPixel3(dest, i, j, &c);
		}
	}
	__syncthreads();
}
__device__ void gaussianPyramid_fast(Pyramid d_outPyr, Image3 *d_inImg, uint8_t nLevels, Kernel d_filter, Pixel3 *ds_downsampled){
	d_imgcpy3(d_outPyr[0], d_inImg);
	uint32_t width = d_inImg->width, height = d_inImg->height;
	//if(0 <= nLevels){ //So it don't need to copy two times the whole img
		downsampleConvolve_fast(d_outPyr[1], d_inImg, &width, &height, d_filter, ds_downsampled);
	//}
	for(uint8_t i = 1; i < nLevels; i++)
		downsampleConvolve_fast(d_outPyr[i + 1], d_outPyr[i], &width, &height, d_filter, ds_downsampled);
	//No extra synchtreads needed because there already is one at the end of downsampleConvolve 
}
__global__ void gaussianPyramid_fastTest(Pyramid d_outPyr, Image3 *d_inImg, uint8_t nLevels, Kernel d_filter){
	__shared__ Pixel3 convolveWorkingBuffer[MAX_PYR_LAYER * MAX_PYR_LAYER];
	gaussianPyramid_fast(d_outPyr, d_inImg, nLevels, d_filter, convolveWorkingBuffer);
}

__device__ void upsampleConvolve(Image3 *dest, Image3 *source, Kernel kernel){
	uint32_t smallWidth = source->width, smallHeight = source->height;
	uint32_t uppedW = smallWidth << 1;
	uint32_t uppedH = smallHeight << 1;
	if(threadIdx.x == 0){
		dest->width = uppedW;
		dest->height = uppedH;
	}
	//__syncthreads();
	const uint8_t  rows = KERNEL_DIMENSION;
	const uint8_t  cols = KERNEL_DIMENSION;
	const int32_t  xstart = -1 * cols / 2;
	const int32_t  ystart = -1 * rows / 2;
	Pixel3 *srcPx = source->pixels;
	Pixel3 *dstPx = dest->pixels;
	
	uint32_t dim = uppedW * uppedH;
	uint32_t max = dim / blockDim.x;
	for(uint32_t li = 0; li <= max; li++){
		uint32_t idx = li * blockDim.x + threadIdx.x;
		if(idx < dim){
			uint32_t i = idx % uppedW, j = idx / uppedW;

			Pixel3 c = zero3vect;
			for (uint32_t y = 0; y < rows; y++) {
                int32_t jy = (j + ystart + y) / 2;
				for (uint32_t x = 0; x < cols; x++) {
                    int32_t ix = (i + xstart + x) / 2;

					int32_t oob = ix >= 0 && ix < smallWidth && jy >= 0 && jy < smallHeight;
					int32_t fi = ix * oob + (i / 2) * (1 - oob), fj = jy * oob + (j / 2) * (1 - oob);

					float kern_elem = kernel[getKernelPosition(x, y)];
					Pixel3 px = d_getPixel3(srcPx, smallWidth, fi, fj); //srcPx[fj * smallWidth + fi];
					c.x += px.x * kern_elem;
					c.y += px.y * kern_elem;
					c.z += px.z * kern_elem;
				}
			}
			d_setPixel3(dstPx, uppedW, i, j, c);
		}
	}
	__syncthreads();
}

__device__ void downsampleConvolve(Image3 *dest, Image3 *source, uint32_t *width, uint32_t *height, Kernel filter){
	const uint32_t originalW = *width, originalH = *height;
	const uint32_t downW = originalW / 2, downH = originalH / 2;
	//printf("a\n");
	//printf("Addr: *width: 0x%016llx  *height: 0x%016llx  *dW: 0x%016llx  *dH: 0x%016llx\n", width, height, &(dest->width), &(dest->height));
	//printf("Entering downsampleConvolve Tid: %d   orgW: %d   orgH: %d   downW: %d   downH: %d - ptrW: %d   ptrH: %d   dW: %d   dH: %d\n", threadIdx.x, originalW, originalH, downW, downH, *width, *height, dest->width, dest->height);
	*width = downW;
	*height = downH;
	if(threadIdx.x == 0){
		dest->width = downW;
		dest->height = downH;
	}
	__syncthreads();
	//printf("Resized dimensions: Tid: %d   orgW: %d   orgH: %d   downW: %d   downH: %d - ptrW: %d   ptrH: %d   dW: %d   dH: %d\n", threadIdx.x, originalW, originalH, downW, downH, *width, *height, dest->width, dest->height);
	const int32_t startingX = originalW & 1;
	const int32_t startingY = originalH & 1;
	const int8_t  rows = KERNEL_DIMENSION;
	const int8_t  cols = KERNEL_DIMENSION;
	const int32_t  xstart = -1 * cols / 2;
	const int32_t  ystart = -1 * rows / 2;
	Pixel3 *srcPx = source->pixels;
	Pixel3 *dstPx = dest->pixels;
	//printf("b\n");

	const int32_t dim = downW * downH; //Small dimensions
	const int32_t max = dim / blockDim.x;
	//printf("Entering loop Tid: %d\n", threadIdx.x);
	for(uint32_t li = 0; li <= max; li++){
		int32_t idx = li * blockDim.x + threadIdx.x;
		int32_t i = (idx % downW) * 2 + startingX, j = (idx / downW) * 2 + startingY;
		//printf("IDX: % 6d            i:% 4d            j:% 4d            im:% 4d            jd:% 4d            downW: %d   downH: %d   orgW: %d   orgH: %d   bool: %u%u\n", idx, i, j, (idx % downW), (idx / downW), downW, downH, originalW, originalH, i < originalH, );
		//if(threadIdx.x == 1 && li > 53300) printf("[%d; %d * %d + %d] Starting loop j: %d   i: %d   originalW: %d   originalH: %d   downW: %d   downH: %d\n", idx, li, blockDim.x, threadIdx.x, originalW, originalH, downW, downH);
		if(i < originalW && j < originalH){

	//for (uint32_t j = startingY; j < originalH; j += 2) {
	//	for (uint32_t i = startingX; i < originalW; i += 2) {

			Pixel3 c = zero3vect;
			for (uint32_t y = 0; y < rows; y++) {
				int32_t jy = j + (ystart + y) * 2 - startingY;
				for (uint32_t x = 0; x < cols; x++) {
					int32_t ix = i + (xstart + x) * 2 - startingX;

					int32_t oob = ix >= 0 && ix < originalW && jy >= 0 && jy < originalH;
					int32_t fi = ix * oob + (i - startingX) * (1 - oob), fj = jy * oob + (j - startingY) * (1 - oob);

					float kern_elem = filter[getKernelPosition(x, y)];
					//if(threadIdx.x == 1 && li > 53300) printf("[%d; %d * %d + %d] -> Fi: %d   Fj: %d   originalW: %d   originalH: %d   oob: %d   ix: %d   i: %d   startingX: %d   jy: %d   j: %d   startingY: %d\n", idx, li, blockDim.x, threadIdx.x, fi, fj, originalW, originalH, oob, ix, i, startingX, jy, j, startingY);
					Pixel3 px = d_getPixel3(srcPx, originalW, fi, fj); //srcPx[fj * originalW + fi];
					c.x += px.x * kern_elem;
					c.y += px.y * kern_elem;
					c.z += px.z * kern_elem;

					/*int32_t ix = i + (xstart + x) * 2 - startingX;

					if (ix >= 0 && ix < originalW && jy >= 0 && jy < originalH) {
						float kern_elem = filter[getKernelPosition(x, y)];
						Pixel3 px = d_getPixel3(srcPx, originalW, ix, jy);

						c.x += px.x * kern_elem;
						c.y += px.y * kern_elem;
						c.z += px.z * kern_elem;
					} else {
						
						float kern_elem = filter[getKernelPosition(x, y)];
						//printf("[%d; %d * %d + %d] -> i: %d  j: %d  sX: %d  sY: %d  orgW: %d  orgH: %d  calc: %d  Addr: 0x%016llx\n", idx, li, blockDim.x, threadIdx.x, i, j, startingX, startingY, originalW, originalH, (j - startingY) * originalW + (i - startingX), srcPx);
						Pixel3 px = srcPx[(j - startingY) * (originalW) + (i - startingX)]; //d_getPixel3(srcPx, originalW, i - startingX, j - startingY);

						c.x += px.x * kern_elem;
						c.y += px.y * kern_elem;
						c.z += px.z * kern_elem;
					}*/
				}
			}
			//c.x = 0; c.y = 0; c.z = 0;
			//if(threadIdx.x == 1 && li > 53300) printf("[%d; %d * %d + %d] <- j: %d   i: %d   j2: %d   i2: %d   originalW: %d   originalH: %d\n", idx, li, blockDim.x, threadIdx.x, j, i, j/2, i/2, originalW, originalH);
			d_setPixel3(dstPx, downW, i / 2, j / 2, c);
		}
	}
	//printf("Exiting loop Tid: %d\n", threadIdx.x);
	__syncthreads();
}

__global__ void gaussianPyramid(Pyramid d_outPyr, Image3 *d_inImg, uint8_t nLevels, Kernel d_filter){
	//printf("Tid: %d\n", threadIdx.x);
	__gaussianPyramid_internal(d_outPyr, d_inImg, nLevels, d_filter);
	//printf("gaussianPyramid done\n");
}
__device__ void __gaussianPyramid_internal(Pyramid d_outPyr, Image3 *d_inImg, uint8_t nLevels, Kernel d_filter){
	d_imgcpy3(d_outPyr[0], d_inImg);
	uint32_t width = d_inImg->width, height = d_inImg->height;
	//if(0 <= nLevels){ //So it don't need to copy two times the whole img
		downsampleConvolve(d_outPyr[1], d_inImg, &width, &height, d_filter);
		//Image3 *h_imgDst = d_outPyr[1];
		//printf("%f %f %f %f %f\n", h_imgDst->pixels[0].y, h_imgDst->pixels[0].y, h_imgDst->pixels[1].y, h_imgDst->pixels[2].y, h_imgDst->pixels[3].y, h_imgDst->pixels[4].y);
	//}
	for(uint8_t i = 1; i < nLevels; i++)
		downsampleConvolve(d_outPyr[i + 1], d_outPyr[i], &width, &height, d_filter);
	//No extra synchtreads needed because there already is one at the end of downsampleConvolve 
}

__global__ void laplacianPyramidTest(Pyramid laplacian, Pyramid tempGauss, uint8_t nLevels, Kernel filter){
	laplacianPyramid(laplacian, tempGauss, nLevels, filter);
}
__device__ void laplacianPyramid(Pyramid laplacian, Pyramid tempGauss, uint8_t nLevels, Kernel filter){
	for(uint8_t i = 0; i < nLevels; i++){
		Image3 *upsampled = laplacian[i];
		upsampleConvolve(upsampled, tempGauss[i + 1], filter);
		//No extra synchtreads needed because there already is one at the end of upsampleConvolve 

		Image3 *current = tempGauss[i];
		//TODO Check if min macro works fine for cuda
		Pixel3 *currentPx = current->pixels, *upsampledPx = upsampled->pixels;
		uint32_t yEnd = min(current->height, upsampled->height);
		uint32_t xEnd = min(current->width, upsampled->width);
		uint32_t dim = xEnd * yEnd;
		uint32_t max = dim / blockDim.x;
		for(uint32_t li = 0; li <= max; li++){
			uint32_t idx = li * blockDim.x + threadIdx.x;
			if(idx < dim){
				uint32_t x = idx % xEnd, y = idx / xEnd;
				Pixel3 ups = d_getPixel3(upsampledPx, upsampled->width, x, y);
				Pixel3 crr = d_getPixel3(currentPx, current->width, x, y);
				d_setPixel3(upsampledPx, upsampled->width, x, y, vec3Sub(crr, ups, Pixel3));
				/*Pixel3 ups = upsampledPx[idx];
				Pixel3 crr = currentPx[idx];

				upsampledPx[idx] = vec3Sub(crr, ups, Pixel3);*/
			}
		}
		//__syncthreads();
	}
	//No extra synchtreads needed
	d_imgcpy3(laplacian[nLevels], tempGauss[nLevels]);
}

__global__ void collapse(Image3 *dest, Pyramid laplacianPyr, uint8_t nLevels, Kernel filter){
	__shared__ float lcl_filter[KERNEL_DIMENSION * KERNEL_DIMENSION];
	uint32_t dim = KERNEL_DIMENSION * KERNEL_DIMENSION;
	uint32_t max = dim / blockDim.x;
	for(uint32_t i = 0; i <= max; i++){
		uint32_t idx = i * blockDim.x + threadIdx.x;
		if(idx < dim)
			lcl_filter[idx] = filter[idx];
	}
	__syncthreads();

	Pixel3 *destPxs = dest->pixels;
	for(int8_t lev = nLevels; lev > 1; lev--){ //Using dest as a temp buffer
		Image3 *currentLevel = laplacianPyr[lev], *biggerLevel = laplacianPyr[lev - 1];
		Pixel3 *biggerLevelPxs = biggerLevel->pixels;

		upsampleConvolve(dest, currentLevel, lcl_filter);
		//No extra synchtreads needed because there already is one at the end of upsampleConvolve 
		uint32_t sizeUpsampled = min(dest->width, biggerLevel->width) * min(dest->height, biggerLevel->height);
		uint32_t max = sizeUpsampled / blockDim.x;
		for(uint32_t i = 0; i <= max; i++){
			uint32_t px = i * blockDim.x + threadIdx.x;
			if(px < sizeUpsampled)
				biggerLevelPxs[px] = vec3Add(destPxs[px], biggerLevelPxs[px], Pixel3);
		}
		if(threadIdx.x == 0){
			biggerLevel->width = dest->width;
			biggerLevel->height = dest->height; //This could cause disalignment problem
		}
		__syncthreads();
	}
	//No extra synchtreads needed
	Image3 *currentLevel = laplacianPyr[1], *biggerLevel = laplacianPyr[0];
	Pixel3 *biggerLevelPxs = biggerLevel->pixels;

	upsampleConvolve(dest, currentLevel, lcl_filter);
	uint32_t sizeUpsampled = min(dest->width, biggerLevel->width) * min(dest->height, biggerLevel->height);
	max = sizeUpsampled / blockDim.x;
	for(uint32_t i = 0; i <= max; i++){
		uint32_t px = i * blockDim.x + threadIdx.x;
		if(px < sizeUpsampled)
			destPxs[px] = vec3Add(destPxs[px], biggerLevelPxs[px], Pixel3);
	}
	__syncthreads();
}

#if SYNC_PRIMITIVES_SUPPORTED
__global__ void __d_llf_internal(Pyramid outputLaplacian, Pyramid gaussPyramid, Image3 *img, uint32_t width, uint32_t height, uint8_t lev, uint32_t subregionDimension, Kernel filter, float sigma, float alpha, float beta, PyrBuffer *buffer){
#else
__global__ void __d_llf_internal(Pyramid outputLaplacian, Pyramid gaussPyramid, Image3 *img, uint32_t width, uint32_t height, uint8_t lev, uint32_t subregionDimension, Kernel filter, float sigma, float alpha, float beta, PyrBuffer *buffer, uint16_t elementsNo){
#endif
	__shared__ Pyramid bufferLaplacianPyramid, bufferGaussPyramid;
	__shared__ Pixel3 g0;
	__shared__ NodeBuffer *node;
	__shared__ float lcl_filter[KERNEL_DIMENSION * KERNEL_DIMENSION];
	__shared__ Pixel3 convolveWorkingBuffer[MAX_PYR_LAYER * MAX_PYR_LAYER];

	//printf("Copying blur kernel %ux%u\n", x, y);
	uint32_t dim = KERNEL_DIMENSION * KERNEL_DIMENSION;
	uint32_t max = dim / blockDim.x;
	for(uint32_t i = 0; i <= max; i++){
		uint32_t idx = i * blockDim.x + threadIdx.x;
		if(idx < dim)
			lcl_filter[idx] = filter[idx];
	}
	__syncthreads();

	Image3 *currentGaussLevel = gaussPyramid[lev];
	Image3 *outLev, *crtLev;

	#if SYNC_PRIMITIVES_SUPPORTED
		uint32_t x = blockIdx.x, y = blockIdx.y;
	#else
		if(threadIdx.x == 0){
			//printf("Shared filter: 0x%012llx        Shared convolve buffer: 0x%012llx\n", lcl_filter, convolveWorkingBuffer);
			node = d_aquireBuffer(buffer);
			bufferLaplacianPyramid = node->bufferLaplacianPyramid;
			bufferGaussPyramid = node->bufferGaussPyramid;
			outLev = outputLaplacian[lev];
			crtLev = bufferLaplacianPyramid[lev];
		}
		
		uint32_t currentW = currentGaussLevel->width, currentH = currentGaussLevel->height;
		uint32_t exDim = currentW * currentH;
		uint32_t exMax = exDim / elementsNo;
		for(uint32_t exId = 0; exId <= exMax; exId++){
			uint32_t exIdx = exId * elementsNo + blockIdx.x;
			if(exIdx >= exDim) return;
			uint32_t x = exIdx % currentW, y = exIdx / currentW;
			//printf("llf_internal: block% 4d    thread% 4d    exId% 4d\n", blockIdx.x, threadIdx.x, exIdx);
	#endif

	int32_t full_res_y = (1 << lev) * y;
	int32_t roi_y0 = full_res_y - subregionDimension;
	int32_t roi_y1 = full_res_y + subregionDimension + 1;
	int32_t base_y = max(0, roi_y0);
	int32_t end_y = min(roi_y1, height);
	int32_t full_res_roi_y = full_res_y - base_y;
	int32_t full_res_roi_yShifted = full_res_roi_y >> lev;

	int32_t full_res_x = (1 << lev) * x;
	int32_t roi_x0 = full_res_x - subregionDimension;
	int32_t roi_x1 = full_res_x + subregionDimension + 1;
	int32_t base_x = max(0, roi_x0);
	int32_t end_x = min(roi_x1, width);
	int32_t full_res_roi_x = full_res_x - base_x;

	if(threadIdx.x == 0){
		#if SYNC_PRIMITIVES_SUPPORTED
			node = d_aquireBuffer(buffer);
			bufferLaplacianPyramid = node->bufferLaplacianPyramid;
			bufferGaussPyramid = node->bufferGaussPyramid;
			outLev = outputLaplacian[lev];
			crtLev = bufferLaplacianPyramid[lev];
		#endif 

		g0 = d_getPixel3(currentGaussLevel->pixels, currentGaussLevel->width, x, y);
	}
	__syncthreads();

	//d_subimage3(bufferLaplacianPyramid[0], img, base_x, end_x, base_y, end_y); //Using bufferLaplacianPyramid[0] as temp buffer
	//d_remap(bufferLaplacianPyramid[0], g0, sigma, alpha, beta);
	d_subimage3Remap(bufferLaplacianPyramid[0], img, base_x, end_x, base_y, end_y, g0, sigma, alpha, beta);
	uint8_t currentNLevels = lev + 1;
	gaussianPyramid_fast(bufferGaussPyramid, bufferLaplacianPyramid[0], currentNLevels, lcl_filter, convolveWorkingBuffer);
	//upsampleConvolveSubtract_fast(bufferLaplacianPyramid[lev], bufferGaussPyramid[currentNLevels], bufferGaussPyramid[lev], lcl_filter, convolveWorkingBuffer);
	Pixel3 outPx = upsampleConvolveSubtractSinglePixel(bufferGaussPyramid[currentNLevels], d_getPixel3(bufferGaussPyramid[lev]->pixels, bufferGaussPyramid[lev]->width, full_res_roi_x >> lev, full_res_roi_yShifted), lcl_filter, full_res_roi_x >> lev, full_res_roi_yShifted);

	if(threadIdx.x == 0){
		d_setPixel3(outLev->pixels, outLev->width, x, y, outPx); //idk why i had to shift those
		//d_setPixel3(outLev->pixels, outLev->width, x, y, d_getPixel3(crtLev->pixels, crtLev->width, full_res_roi_x >> lev, full_res_roi_yShifted)); //idk why i had to shift those
		
		//d_releaseBuffer(node, buffer);
	}
	//__syncthreads();
	
	#if !(SYNC_PRIMITIVES_SUPPORTED)
		}
	#endif
}

__host__ void llf(Image3 *h_img, float h_sigma, float h_alpha, float h_beta, uint8_t h_nLevels, uint32_t h_nThreads, uint32_t h_elementsNo){

	size_t rsize = 1024ULL*1024ULL*1024ULL*4ULL;  // allocate 4GB
	CHECK(hipDeviceSetLimit(hipLimitMallocHeapSize, rsize));

	struct timeval start, stop;
	uint64_t passed = 0;

	uint32_t h_width = h_img->width, h_height = h_img->height;
	h_nLevels = min(h_nLevels, MAX_LAYERS);
	h_nLevels = max(h_nLevels, 3);//int(ceil(std::abs(std::log2(min(width, height)) - 3))) + 2;
	print("Creating blur kernel");
	Kernel d_filter = createFilterDevice();
	print("Creating gauss pyr");
	Pyramid d_gaussPyramid = createPyramidDevice(h_width, h_height, h_nLevels);
	print("Creating lapl pyr");
	Pyramid d_outputLaplacian = createPyramidDevice(h_width, h_height, h_nLevels);

	print("Create buffer device");
	PyrBuffer *d_buffer = createBufferDevice(h_elementsNo, (3 * ((1 << (h_nLevels + 1)) - 1)), h_nLevels);

	print("makeimage3");
	Image3 *d_img = makeImage3Device(h_width, h_height);
	print("copyImg3");
	copyImg3Host2Device(d_img, h_img);
	print("FIRST KERNEL");
	gettimeofday(&start, NULL);
	gaussianPyramid<<<1, h_nThreads>>>(d_gaussPyramid, d_img, h_nLevels, d_filter);
	CHECK(hipDeviceSynchronize());
	gettimeofday(&stop, NULL);
	//passed = (stop.tv_sec - start.tv_sec) * 1000000 + stop.tv_usec - start.tv_usec;

	/*//Image3 *d_tempImg = getImageFromPyramidDevice(d_outputLaplacian, 0);
	//const uint32_t x = 368, y = 279;
	//h_width = 200; h_height = 200;
	////d_subimage3Test<<<1, 128>>>(d_tempImg, d_img, x, x + h_width, y, y + h_height);	
	//gaussianPyramid<<<1, 128>>>(d_gaussPyramid, d_tempImg, h_nLevels, d_filter);
	Image3 *d_blurImg = getImageFromPyramidDevice(d_gaussPyramid, 0);
	//d_clampImage3<<<(((h_width * h_height) + h_nThreads - 1) / h_nThreads), h_nThreads>>>(d_blurImg);
	copyImg3Device2Host(h_img, d_blurImg);
	/*laplacianPyramidTest<<<1, 128>>>(d_outputLaplacian, d_gaussPyramid, h_nLevels, d_filter);
	CHECK(hipDeviceSynchronize());
	d_copyPyrLevel<<<1, 128>>>(d_outputLaplacian, d_gaussPyramid, h_nLevels);
	CHECK(hipDeviceSynchronize());
	collapse<<<1, 128>>>(d_img, d_outputLaplacian, h_nLevels, d_filter);
	CHECK(hipDeviceSynchronize());

	fflush(stdout);

	Image3 *d_blurImg = getImageFromPyramidDevice(d_gaussPyramid, 0);
	//d_clampImage3<<<(((h_width * h_height) + h_nThreads - 1) / h_nThreads), h_nThreads>>>(d_img);
	//CHECK(hipDeviceSynchronize());
	//d_subimage3Test<<<1, 64>>>(d_img, d_blurImg, 500, 625, 32, 190);
	copyImg3Device2Host(h_img, d_blurImg);*/

	gettimeofday(&start, NULL);
	for(uint8_t h_lev = 0; h_lev < h_nLevels; h_lev++){
		printff("Loop %u\n", h_lev);
		uint32_t h_subregionDimension = 3 * ((1 << (h_lev + 2)) - 1) / 2;

		#if SYNC_PRIMITIVES_SUPPORTED
			uint32_t h_layerW, h_layerH;
			getPyramidDimensionsAtLayer(d_gaussPyramid, h_lev, &h_layerW, &h_layerH);
			dim3 grid(h_layerW, h_layerH);
			__d_llf_internal<<<grid, h_nThreads>>>(d_outputLaplacian, d_gaussPyramid, d_img, h_width, h_height, h_lev, h_subregionDimension, d_filter, h_sigma, h_alpha, h_beta, d_buffer);
		#else
			//h_elementsNo, h_nThreads
			__d_llf_internal<<<h_elementsNo, h_nThreads>>>(d_outputLaplacian, d_gaussPyramid, d_img, h_width, h_height, h_lev, h_subregionDimension, d_filter, h_sigma, h_alpha, h_beta, d_buffer, h_elementsNo);
		#endif
		CHECK(hipDeviceSynchronize());
		//break;
	}
	gettimeofday(&stop, NULL);
	d_copyPyrLevel<<<1, h_nThreads>>>(d_outputLaplacian, d_gaussPyramid, h_nLevels);
	CHECK(hipDeviceSynchronize());
	collapse<<<1, h_nThreads>>>(d_img, d_outputLaplacian, h_nLevels, d_filter);
	CHECK(hipDeviceSynchronize());
	passed += (stop.tv_sec - start.tv_sec) * 1000000 + stop.tv_usec - start.tv_usec;
	passed /= 1000;
	printff("Total time: %lums\n", passed);

	d_clampImage3<<<(((h_width * h_height) + h_nThreads - 1) / h_nThreads), h_nThreads>>>(d_img);
	CHECK(hipDeviceSynchronize());

	copyImg3Device2Host(h_img, d_img);

	destroyBufferDevice(h_elementsNo, h_nLevels, d_buffer);
	destroyImage3Device(d_img);
	destroyPyramidDevice(d_gaussPyramid, h_nLevels);
	destroyPyramidDevice(d_outputLaplacian, h_nLevels);
	destroyFilterDevice(d_filter);
}

uint32_t getPixelNoPerPyramid(uint8_t nLevels){
	uint32_t subregionDimension = 3 * ((1 << (nLevels + 2)) - 1);
	uint32_t totalPixels = 0;
	for(uint8_t i = 0; i <= nLevels; i++){
		totalPixels += (subregionDimension * subregionDimension);
		subregionDimension = subregionDimension / 2 + (subregionDimension & 1);
	}
	return totalPixels;
}

int main(){
	Image4 *img4 = getStaticImage4();
	Image3 *img = image4to3(img4);
	AlphaMap map = getAlphaMap(img4);
	destroyImage4(&img4);

	llf(img, 0.35, 0.4, 5, 3, 640, 128);

	img4 = image3to4AlphaMap(img, map);
	destroyImage3(&img);
	printStaticImage4(img4);
	destroyImage4(&img4);
}